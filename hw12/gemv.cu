#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h> 


#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

// if N>= 128
__global__ void Sgemv_v1(
    float* __restrict__ A,
    float* __restrict__ x,
    float* __restrict__ y,
    const int M,
    const int N) {
    // Block index
    int bx = blockIdx.x;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int warp_size = 32;
    int laneId = tx % warp_size;
    int current_row = blockDim.y * bx + ty;

    if (current_row < M) {
        float res = 0;
        int kIteration = (N / warp_size) / 4;
        if (kIteration == 0) kIteration = 1;
        A = &A[current_row * N];
#pragma unroll
        for (int i = 0; i < kIteration; i++) {
            int current_col_vec = (i * warp_size + laneId);
            float4 current_val = reinterpret_cast<float4*>(A)[current_col_vec];
            float4 current_x = reinterpret_cast<float4*>(x)[current_col_vec];
            res += current_val.x * current_x.x;
            res += current_val.y * current_x.y;
            res += current_val.z * current_x.z;
            res += current_val.w * current_x.w;
        }
        res = warpReduceSum<warp_size>(res);
        if (laneId == 0) y[current_row] = res;
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        printf("usage: ./main [M] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t N = atoi(argv[2]);

    size_t bytes_A = sizeof(float) * M * N;
    size_t bytes_x = sizeof(float) * N;
    size_t bytes_y = sizeof(float) * M;
    float* h_A = (float*)malloc(bytes_A);
    float* h_x = (float*)malloc(bytes_x);
    float* h_y = (float*)malloc(bytes_y);
    float* h_y1 = (float*)malloc(bytes_y);

    float* d_A;
    float* d_x;
    float* d_y;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_x, bytes_x));
    checkCudaErrors(hipMalloc(&d_y, bytes_y));

    // 生成A的数据
    for (int i = 0; i < M * N; i++) {
        h_A[i] = (float)i / N;
    }

    // 生成x的数据
    for (int i = 0; i < N; i++) {
        h_x[i] = 1;
    }
    memset(h_y, 0, M * sizeof(float));
    memset(h_y1, 0, M * sizeof(float));

    int nIter = 1000;
    checkCudaErrors(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, h_y, bytes_y, hipMemcpyHostToDevice));
    for (int run = 0; run < nIter; run++) {
        dim3 dimGrid((M + 3) / 4);
        dim3 dimBlock(32, 4);
        Sgemv_v1 << < dimGrid, dimBlock >> > (d_A, d_x, d_y, M, N);
    }
    checkCudaErrors(hipMemcpy(h_y, d_y, bytes_y, hipMemcpyDeviceToHost));

    // cublas
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy(d_y, h_y1, bytes_y, hipMemcpyHostToDevice));
    for (int run = 0; run < nIter; run++) {
        hipblasSgemv(blas_handle, HIPBLAS_OP_T,
            N, M, &alpha,
            d_A, N, d_x, 1, &beta, d_y, 1
        );
    }
    checkCudaErrors(hipMemcpy(h_y1, d_y, bytes_y, hipMemcpyDeviceToHost));
    hipblasDestroy(blas_handle);

    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M; i++) {
        double abs_err = fabs(h_y[i] - h_y1[i]);
        double dot_length = M;
        double abs_val = fabs(h_y[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                i, h_y[i], h_y1[i], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");

    // Free Memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    free(h_A);
    free(h_x);
    free(h_y);
    free(h_y1);
}
