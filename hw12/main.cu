#include "hip/hip_runtime.h"
// 这是程序二的模板程序，我们已经准备好了加载数据集和加载程序一模型参数的部分，请实现CUDA的深度学习推理过程，请严格保持输出格式输出
// 编译的命令为：nvcc test.cu -o test -Xcompiler "-O3 -std=c++14" -gencode arch=compute_50,code=sm_50 -gencode arch=compute_52,code=sm_52 -gencode arch=compute_53,code=sm_53 -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_62,code=sm_62 -gencode arch=compute_70,code=sm_70

#include <fstream>
#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <string>

#include <hip/hip_runtime.h>

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}
// #define DEBUG
#define element_type float
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])



// 读取MNIST数据集
std::vector<float> read_mnist_images(const std::string& path) {
    std::ifstream file(path, std::ios::binary);
    if (!file) {
        std::cout << "Cannot open file!" << path << std::endl;
        return {};
    }

    int magic_number = 0, num_images = 0, num_rows = 0, num_cols = 0;
    file.read((char*)&magic_number, sizeof(magic_number));
    file.read((char*)&num_images, sizeof(num_images));
    file.read((char*)&num_rows, sizeof(num_rows));
    file.read((char*)&num_cols, sizeof(num_cols));

    // Reverse Integers (MNIST data is in big endian format)
    magic_number = ((magic_number & 0xff000000) >> 24) | ((magic_number & 0x00ff0000) >> 8) |
        ((magic_number & 0x0000ff00) << 8) | ((magic_number & 0x000000ff) << 24);
    num_images = ((num_images & 0xff000000) >> 24) | ((num_images & 0x00ff0000) >> 8) |
        ((num_images & 0x0000ff00) << 8) | ((num_images & 0x000000ff) << 24);
    num_rows = ((num_rows & 0xff000000) >> 24) | ((num_rows & 0x00ff0000) >> 8) |
        ((num_rows & 0x0000ff00) << 8) | ((num_rows & 0x000000ff) << 24);
    num_cols = ((num_cols & 0xff000000) >> 24) | ((num_cols & 0x00ff0000) >> 8) |
        ((num_cols & 0x0000ff00) << 8) | ((num_cols & 0x000000ff) << 24);
    // std::cout << "magic_number " << magic_number << "num_images" << num_images << "num_rows" << num_rows << "num_cols" << num_cols << std::endl;
    int image_size = num_rows * num_cols;
    std::vector<float> images(num_images * 28 * 28, 0);

    for (int i = 0; i < num_images; ++i) {
        for (int j = 0; j < image_size; ++j) {
            unsigned char pixel = 0;
            file.read((char*)&pixel, sizeof(pixel));

            images[i * 28 * 28 + j] = static_cast<float>(pixel) / 255.0f;
            images[i * 28 * 28 + j] = 2 * images[i * 28 * 28 + j] - 1;
            // if (i == 0)
            // {
            //     std::cout << static_cast<float>(pixel) << " ";
            // }
        }
    }

    return images;
}

// 读取MNIST label数据集
std::vector<int> read_mnist_labels(const std::string& path) {
    std::ifstream file(path, std::ios::binary);
    if (!file) {
        std::cout << "Cannot open file!" << path << std::endl;
        return {};
    }

    int magic_number = 0, num_items = 0;
    file.read((char*)&magic_number, sizeof(magic_number));
    file.read((char*)&num_items, sizeof(num_items));

    // Reverse Integers (MNIST data is in big endian format)
    magic_number = ((magic_number & 0xff000000) >> 24) | ((magic_number & 0x00ff0000) >> 8) |
        ((magic_number & 0x0000ff00) << 8) | ((magic_number & 0x000000ff) << 24);
    num_items = ((num_items & 0xff000000) >> 24) | ((num_items & 0x00ff0000) >> 8) |
        ((num_items & 0x0000ff00) << 8) | ((num_items & 0x000000ff) << 24);

    std::vector<int> labels(num_items);
    for (int i = 0; i < num_items; ++i) {
        unsigned char label = 0;
        file.read((char*)&label, sizeof(label));
        labels[i] = static_cast<int>(label);
    }

    return labels;
}

// 读取模型参数
std::vector<float> read_param(const std::string& path) {
    std::ifstream file(path);
    std::vector<float> params;
    float param;
    while (file >> param) {
        params.push_back(param);
    }
    return params;
}

// 范例kernel函数，无实际作用
__global__ void add_arrays(int* a, int* b, int* c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

void printTensor(std::vector<float> A, int rowS, int colS, int chaS)
{
    for (int c = 0; c < chaS; c++)
    {

        std::cout << "channel : " << c << std::endl;
        for (int i = 0; i < rowS; i++)
        {

            for (int j = 0; j < colS; j++)
            {
                std::cout << A[c * rowS * colS + i * colS + j] << " ";
            }
            std::cout << std::endl;
        }
    }

}

template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}


__device__ void warpReduce(volatile float* cache, unsigned int tid) {
    cache[tid] += cache[tid + 32];
    cache[tid] += cache[tid + 16];
    cache[tid] += cache[tid + 8];
    cache[tid] += cache[tid + 4];
    cache[tid] += cache[tid + 2];
    cache[tid] += cache[tid + 1];
}

__global__ void reduce(int* predict, int* labels, int* sum, int N) {
    __shared__ float sdata[256];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // printf("predict[i] =%d  predict[i + blockDim.x] %d\n", predict[i], predict[i + blockDim.x]);
    if (i < N)
        sdata[tid] = (predict[i] == labels[i]);
    if (i + blockDim.x < N)
        sdata[tid] += (predict[i + blockDim.x] == labels[i + blockDim.x]);
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid < 32) warpReduce(sdata, tid);
    if (tid == 0) sum[blockIdx.x] = sdata[0];
}

int check(int* d_predict, int* d_labels, int N)
{
    int THREAD_PER_BLOCK = 256;
    int NUM_PER_BLOCK = 2 * 256;
    // printf("N %d\n", N);
    int block_num = (N + NUM_PER_BLOCK - 1) / NUM_PER_BLOCK;
    int* d_sum, * sum = (int*)malloc(sizeof(int) * block_num);

    // printf("N %d. block_num %d\n", N, block_num);
    hipMalloc(&d_sum, block_num * sizeof(int));

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    reduce << <Grid, Block >> > (d_predict, d_labels, d_sum, N);
    hipMemcpy(sum, d_sum, block_num * sizeof(int), hipMemcpyDeviceToHost);
    int ans = 0;
    for (int i = 0; i < block_num; i++)
    {
        ans += sum[i];
        // printf("sum[%d] = %d\n", i, sum[i]);
    }
    return ans;
    hipFree(d_sum);

}
void init_ij(std::vector<float>& A, int n, int m, int c)
{
    for (int i = 0; i < c; i++)
        for (int j = 0; j < n; j++)
        {
            for (int k = 0; k < m; k++)
                A[i * n * m + j * m + k] = k + j;
        }
}



__device__ void print_d(float* y, int len)
{
    for (int i = 0; i < len; i++)
        printf("%f\n", y[i]);
}

template<int set_size>
__global__ void _lenet_fusion_new(float* input, const float* __restrict__ kernel,
    const float* __restrict__ kernel_bias, const float* __restrict__ kernel2,
    const float* __restrict__ kernel_bias2,
    float* __restrict__ A,
    int* labels, int* sum,
    int set_id) //是方形
{
    int t = set_id * set_size + blockIdx.x;
    int flag = labels[t];
    // clock_t start_conv_time = clock();
    input = &input[(t) * 28 * 28];
    int inputChannel, outputChannel, inputSize, kernelSize;
    inputChannel = 1, outputChannel = 6, inputSize = 28, kernelSize = 5;


    __shared__ float in_s[6][28][28];
    __shared__ float in_pool_s[28][28];
    __shared__ float ker_s[16][5][5];

    __shared__ float output_pool[6][12][12];
    __shared__ float output_pool2[16 * 4 * 4];

    int outputSize = inputSize - kernelSize + 1;

    int destY = threadIdx.y, destX = threadIdx.x;
    int srcY = destY, srcX = destX;
    for (int ic = 0; ic < inputChannel; ic++)
    {

        if (threadIdx.y < inputSize && threadIdx.x < inputSize)
        {
            int in_pos = ic * inputSize * inputSize + threadIdx.y * inputSize + threadIdx.x;
            in_s[ic][destY][destX] = input[in_pos];

        }
    }
    // __syncthreads();
    for (int oc = 0; oc < outputChannel; oc++)
    {

        float tmp_bias = kernel_bias[oc];
        float accum = 0;
        for (int ic = 0; ic < inputChannel; ic++)
        {
            if (destY < kernelSize && destX < kernelSize)
            {
                int ker_pos = oc * kernelSize * kernelSize * inputChannel +
                    ic * kernelSize * kernelSize + destY * kernelSize + destX;
                ker_s[ic][destY][destX] = kernel[ker_pos];
            }
        }
        __syncthreads(); //奇怪，这个同步不能去
        for (int ic = 0; ic < inputChannel; ic++)
        {
            if (destY < kernelSize && destX < kernelSize)
            {
                int ker_pos = oc * kernelSize * kernelSize * inputChannel +
                    ic * kernelSize * kernelSize + destY * kernelSize + destX;
                ker_s[ic][destY][destX] = kernel[ker_pos];
            }


            if (srcY + kernelSize - 1 < inputSize && srcX + kernelSize - 1 < inputSize)
            {
                for (int i = 0; i < kernelSize; i++)
                {
#pragma unroll
                    for (int j = 0; j < kernelSize; j++)
                    {
                        accum += in_s[ic][srcY + i][srcX + j] * ker_s[ic][i][j];
                    }
                }

            }
        }



        if (destY < outputSize && destX < outputSize)
            in_pool_s[destY][destX] = accum + tmp_bias;

        __syncthreads();


        int output_pool_size = outputSize / 2;
        int kernel_pool_size = 2;
        if (srcY < output_pool_size && srcX < output_pool_size)
        {
            float tmp_max = 0;
            for (int i = 0; i < kernel_pool_size; i++)
#pragma unroll
                for (int j = 0; j < kernel_pool_size; j++)
                {

                    tmp_max = max(tmp_max, in_pool_s[srcY * kernel_pool_size + i][srcX * kernel_pool_size + j]);
                }
            output_pool[oc][srcY][srcX] = tmp_max >= 0 ? tmp_max : 0;
        }
    }
    __syncthreads();
    //-----------_conv2d_1<2> << < 1, block >> > (d_output_pool, d_output_pool2, d_kernel2, d_kernelBias2);-----------
    //------------------------------------------------second--------------------------------------------------------------
    inputChannel = 6, outputChannel = 16, inputSize = 12, kernelSize = 5;
    outputSize = inputSize - kernelSize + 1;


    for (int oc = 0; oc < outputChannel; oc++)
    {

        float tmp_bias = kernel_bias2[oc];
        float accum = 0;
        for (int ic = 0; ic < inputChannel; ic++)
        {
            if (destY < kernelSize && destX < kernelSize)
            {
                int ker_pos = oc * kernelSize * kernelSize * inputChannel +
                    ic * kernelSize * kernelSize + destY * kernelSize + destX;
                ker_s[ic][destY][destX] = kernel2[ker_pos];
            }
        }
        __syncthreads(); //奇怪，这个同步不能去
        for (int ic = 0; ic < inputChannel; ic++)
        {
            if (srcY + kernelSize - 1 < inputSize && srcX + kernelSize - 1 < inputSize)
            {
                for (int i = 0; i < kernelSize; i++)
                {
#pragma unroll
                    for (int j = 0; j < kernelSize; j++)
                    {
                        accum += output_pool[ic][srcY + i][srcX + j] * ker_s[ic][i][j];
                    }
                }

            }
        }



        if (destY < outputSize && destX < outputSize)
            in_pool_s[destY][destX] = accum + tmp_bias;

        __syncthreads();

        int output_pool_size = outputSize / 2;
        int kernel_pool_size = 2;
        // 4 * 4 * 16
        if (srcY < output_pool_size && srcX < output_pool_size)
        {
            float tmp_max = 0;
            for (int i = 0; i < kernel_pool_size; i++)
#pragma unroll
                for (int j = 0; j < kernel_pool_size; j++)
                {

                    tmp_max = max(tmp_max, in_pool_s[srcY * kernel_pool_size + i][srcX * kernel_pool_size + j]);
                }
            int out_pos = oc * output_pool_size * output_pool_size + srcY * output_pool_size + srcX;
            if (tmp_max >= 0)
            {

                output_pool2[out_pos] = tmp_max;
            }
            else
            {
                output_pool2[out_pos] = 0;
            }
        }
    }
    __syncthreads();

    // clock_t end_conv_time = clock();
    // clock_t start_fc_time = clock();
    //------------------------------------------------relu+gemv--------------------------------------------------------------
    //                                                 GEMV
    //------------------------------------------------relu+gemv--------------------------------------------------------------

    //一个warp算y的一个元素
    int height = 10, width = 256;
    int warp_id = threadIdx.y;
    int warp_num = blockDim.y;
    const int warp_size = 32;

    //warp要取的col的start
    int col_vec_start = threadIdx.x;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    // __shared__ Arow_s[width];  
    float* x_s = output_pool2;
    __shared__ float out[10];




#pragma unroll
    for (int row = warp_id / 2; row < height; row += warp_num)
    {
        out[row] = 0;
        float tmp = 0;
        col_vec_start += warp_id % 2 * 32;
        float4 current_val1 = reinterpret_cast<float4*>(A)[row * width / 4 + col_vec_start];
        tmp += current_val1.x * x_s[col_vec_start * 4];
        tmp += current_val1.y * x_s[col_vec_start * 4 + 1];
        tmp += current_val1.z * x_s[col_vec_start * 4 + 2];
        tmp += current_val1.w * x_s[col_vec_start * 4 + 3];
        tmp = warpReduceSum<warp_size>(tmp);
        if (threadIdx.x == 0)
        {
            atomicAdd(&out[row], tmp);
        }

    }
    __syncthreads();

    if (tid == 0)
    {
        float tmp_max = -1e9, id = 0;
        for (int i = 0; i < 10; i++)
        {
            if (tmp_max < out[i])
            {
                tmp_max = out[i], id = i;
            }
        }
        int acc = (id == flag);
        atomicAdd(sum, acc);
    }
}

int main(int argc, char* argv[]) {
    std::string dir = argv[1];  // 第一个参数是程序所在的目录，这个目录是存放前一步训练模型参数文件的目录，从这个目录下读取模型参数文件，相对于这个目录读取测试集图片和标签
    // cout << dir;

    // 读取测试集，对于想实现CUDA C/C++训练的同学，参考训练集文件名为train-images-idx3-ubyte和train-labels-idx1-ubyte
#if 1
    auto images = read_mnist_images(dir + "/../../data/FashionMNIST/raw/t10k-images-idx3-ubyte");
    // 读取测试集标签
    auto labels = read_mnist_labels(dir + "/../../data/FashionMNIST/raw/t10k-labels-idx1-ubyte");
    // 读取模型参数
    auto conv1_weight = read_param(dir + "/conv1.weight.txt");
    auto conv1_bias = read_param(dir + "/conv1.bias.txt");
    auto conv2_weight = read_param(dir + "/conv2.weight.txt");
    auto conv2_bias = read_param(dir + "/conv2.bias.txt");
    auto fc4_weight = read_param(dir + "/fc1.weight.txt");
    auto fc4_bias = read_param(dir + "/fc1.bias.txt");
    float* d_fc4_weight, * d_fc4_bias;
    // printf("fc4 size %d bias %d\n", fc4_weight.size(), fc4_bias.size());
#else
    auto images = read_mnist_images(dir + "/data/FashionMNIST/raw/t10k-images-idx3-ubyte");
    // 读取测试集标签
    auto labels = read_mnist_labels(dir + "/data/FashionMNIST/raw/t10k-labels-idx1-ubyte");
    // 读取模型参数
    // // std::cout << dir << std::endl;
    auto conv1_weight = read_param(dir + "/conv1.weight_epoch400.txt");
    auto conv1_bias = read_param(dir + "/conv1.bias_epoch400.txt");
    auto conv2_weight = read_param(dir + "/conv2.weight_epoch400.txt");
    auto conv2_bias = read_param(dir + "/conv2.bias_epoch400.txt");
    auto fc1_weight = read_param(dir + "/fc1.weight_epoch400.txt");
    auto fc1_bias = read_param(dir + "/fc1.bias_epoch400.txt");
    auto fc2_weight = read_param(dir + "/fc2.weight_epoch400.txt");
    auto fc2_bias = read_param(dir + "/fc2.bias_epoch400.txt");
    auto fc3_weight = read_param(dir + "/fc3.weight_epoch400.txt");
    auto fc3_bias = read_param(dir + "/fc3.bias_epoch400.txt");
    // auto conv1_weight = read_param(dir + "/conv1.weight.txt");
    // auto conv1_bias = read_param(dir + "/conv1.bias.txt");
    // auto conv2_weight = read_param(dir + "/conv2.weight.txt");
    // auto conv2_bias = read_param(dir + "/conv2.bias.txt");
    // auto fc1_weight = read_param(dir + "/fc1.weight.txt");
    // auto fc1_bias = read_param(dir + "/fc1.bias.txt");
    // auto fc2_weight = read_param(dir + "/fc2.weight.txt");
    // auto fc2_bias = read_param(dir + "/fc2.bias.txt");
    // auto fc3_weight = read_param(dir + "/fc3.weight.txt");
    // auto fc3_bias = read_param(dir + "/fc3.bias.txt");
#endif
    // 打印每一个标签，仅用于调试！

    // for (const auto& label : labels) {
    //     std::cout << label << " ";
    // }
    // std::cout<<std::endl;

    hipFree(0);


    float* d_input;
    float* d_conv1_weight, * d_conv1_bias, * d_conv2_weight, * d_conv2_bias;
    int* d_predict, * d_labels;
    int* predict = (int*)malloc(sizeof(int) * labels.size());
    const int set_size = 10000 / 40;
    int nStreams = 20;
    hipStream_t streams[nStreams];
    for (int i = 0; i < nStreams; i++) {
        hipStreamCreate(&streams[i]);
    }



    hipMalloc(&d_fc4_weight, fc4_weight.size() * sizeof(float));
    hipMalloc(&d_fc4_bias, fc4_bias.size() * sizeof(float));
    hipMemcpy(d_fc4_weight, fc4_weight.data(), sizeof(float) * fc4_weight.size(), hipMemcpyHostToDevice);

    hipMalloc(&d_input, 10000 * 28 * 28 * sizeof(float));
    hipMalloc(&d_predict, sizeof(int) * labels.size());
    hipMalloc(&d_labels, sizeof(int) * labels.size());
    hipMalloc(&d_conv1_weight, conv1_weight.size() * sizeof(float));
    hipMalloc(&d_conv1_bias, conv1_bias.size() * sizeof(float));
    hipMalloc(&d_conv2_weight, conv2_weight.size() * sizeof(float));
    hipMalloc(&d_conv2_bias, conv2_bias.size() * sizeof(float));

    // hipMemcpy(d_input, images.data(), sizeof(float) * images.size(), hipMemcpyHostToDevice);

    hipMemcpy(d_labels, labels.data(), labels.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_weight, conv1_weight.data(), sizeof(float) * conv1_weight.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_bias, conv1_bias.data(), sizeof(float) * conv1_bias.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_weight, conv2_weight.data(), sizeof(float) * conv2_weight.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_bias, conv2_bias.data(), sizeof(float) * conv2_bias.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_input, &images[0],
        sizeof(float) * images.size(), hipMemcpyHostToDevice);

    // int sum = 0;
    
    int N = labels.size();
            int THREAD_PER_BLOCK = 256;
        int NUM_PER_BLOCK = 2 * 256;
        // printf("N %d\n", N);
        int block_num = (N + NUM_PER_BLOCK - 1) / NUM_PER_BLOCK;
        int* d_sum, * sum = (int*)malloc(sizeof(int) * block_num);

        // printf("N %d. block_num %d\n", N, block_num);
        hipMalloc(&d_sum, block_num * sizeof(int));

        dim3 Grid(block_num, 1);
        dim3 Block(THREAD_PER_BLOCK, 1);
    
        auto start = std::chrono::high_resolution_clock::now();
    // 开始计时，使用chrono计时，不支持其它计时方式
        for (int t = 0; t < 10000 / set_size; t++) {
        int stream_tid = t % nStreams;
        dim3 block(32, 32);
        dim3 grid(set_size);

        _lenet_fusion_new<set_size> << < grid, block, 400, streams[stream_tid] >> > (d_input,
            d_conv1_weight,
            d_conv1_bias,
            d_conv2_weight,
            d_conv2_bias,
            d_fc4_weight,
            d_labels,
            d_sum,
            t);

        // hipDeviceSynchronize();

        // std::cout << "real: " << labels[t]<< ", predict : "<<  maxT(output5) << std::endl;
    }

    hipMemcpy(sum, d_sum, block_num * sizeof(int), hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    std::cout << std::fixed << std::setprecision(10) << diff.count() << ":" << std::setprecision(10) << (float)sum[0]/ (float)10000 << std::endl;

    hipMemcpy(sum, d_sum, block_num * sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(predict, d_predict, sizeof(int) *labels.size(), hipMemcpyDeviceToHost);

    hipFree(d_fc4_weight);
    hipFree(d_fc4_bias);
    hipFree(d_conv1_weight);
    hipFree(d_conv1_bias);
    hipFree(d_conv2_weight);
    hipFree(d_conv2_bias);
    hipFree(d_input);
    hipFree(d_predict);
    hipFree(d_labels);
    // 向主机端同步以等待所有异步调用的GPU kernel执行完毕，这句必须要有
    // 结束计时
    // printf("sum = %d\n", sum);

    // 输出结果，请严格保持此输出格式，并把0.0001替换成实际的准确率，请不要输出除了此结果之外的任何内容！！！
    // std::cout << std::fixed << std::setprecision(2) << diff.count() << ":0.0001";

    return 0;
}
