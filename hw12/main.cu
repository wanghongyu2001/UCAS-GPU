#include "hip/hip_runtime.h"
// 这是程序二的模板程序，我们已经准备好了加载数据集和加载程序一模型参数的部分，请实现CUDA的深度学习推理过程，请严格保持输出格式输出
// 编译的命令为：nvcc test.cu -o test -Xcompiler "-O3 -std=c++14" -gencode arch=compute_50,code=sm_50 -gencode arch=compute_52,code=sm_52 -gencode arch=compute_53,code=sm_53 -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_62,code=sm_62 -gencode arch=compute_70,code=sm_70

#include <fstream>
#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <string>

// 读取MNIST数据集
std::vector<std::vector<float>> read_mnist_images(const std::string& path) {
    std::ifstream file(path, std::ios::binary);
    if (!file) {
        std::cout << "Cannot open file!" << path << std::endl;
        return {};
    }

    int magic_number = 0, num_images = 0, num_rows = 0, num_cols = 0;
    file.read((char*)&magic_number, sizeof(magic_number));
    file.read((char*)&num_images, sizeof(num_images));
    file.read((char*)&num_rows, sizeof(num_rows));
    file.read((char*)&num_cols, sizeof(num_cols));

    // Reverse Integers (MNIST data is in big endian format)
    magic_number = ((magic_number & 0xff000000) >> 24) | ((magic_number & 0x00ff0000) >> 8) |
        ((magic_number & 0x0000ff00) << 8) | ((magic_number & 0x000000ff) << 24);
    num_images = ((num_images & 0xff000000) >> 24) | ((num_images & 0x00ff0000) >> 8) |
        ((num_images & 0x0000ff00) << 8) | ((num_images & 0x000000ff) << 24);
    num_rows = ((num_rows & 0xff000000) >> 24) | ((num_rows & 0x00ff0000) >> 8) |
        ((num_rows & 0x0000ff00) << 8) | ((num_rows & 0x000000ff) << 24);
    num_cols = ((num_cols & 0xff000000) >> 24) | ((num_cols & 0x00ff0000) >> 8) |
        ((num_cols & 0x0000ff00) << 8) | ((num_cols & 0x000000ff) << 24);

    int image_size = num_rows * num_cols;
    std::vector<std::vector<float>> images(num_images, std::vector<float>(image_size));

    for (int i = 0; i < num_images; ++i) {
        for (int j = 0; j < image_size; ++j) {
            unsigned char pixel = 0;
            file.read((char*)&pixel, sizeof(pixel));
            images[i][j] = static_cast<float>(pixel) / 255.0f;
        }
    }

    return images;
}

// 读取MNIST label数据集
std::vector<int> read_mnist_labels(const std::string& path) {
    std::ifstream file(path, std::ios::binary);
    if (!file) {
        std::cout << "Cannot open file!" << path << std::endl;
        return {};
    }

    int magic_number = 0, num_items = 0;
    file.read((char*)&magic_number, sizeof(magic_number));
    file.read((char*)&num_items, sizeof(num_items));

    // Reverse Integers (MNIST data is in big endian format)
    magic_number = ((magic_number & 0xff000000) >> 24) | ((magic_number & 0x00ff0000) >> 8) |
        ((magic_number & 0x0000ff00) << 8) | ((magic_number & 0x000000ff) << 24);
    num_items = ((num_items & 0xff000000) >> 24) | ((num_items & 0x00ff0000) >> 8) |
        ((num_items & 0x0000ff00) << 8) | ((num_items & 0x000000ff) << 24);

    std::vector<int> labels(num_items);
    for (int i = 0; i < num_items; ++i) {
        unsigned char label = 0;
        file.read((char*)&label, sizeof(label));
        labels[i] = static_cast<int>(label);
    }

    return labels;
}

// 读取模型参数
std::vector<float> read_param(const std::string& path) {
    std::ifstream file(path);
    std::vector<float> params;
    float param;
    while (file >> param) {
        params.push_back(param);
    }
    return params;
}

// 范例kernel函数，无实际作用
__global__ void add_arrays(int* a, int* b, int* c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

int main(int argc, char* argv[]) {
    std::string dir = argv[1];  // 第一个参数是程序所在的目录，这个目录是存放前一步训练模型参数文件的目录，从这个目录下读取模型参数文件，相对于这个目录读取测试集图片和标签
    // cout << dir;

    // 读取测试集，对于想实现CUDA C/C++训练的同学，参考训练集文件名为train-images-idx3-ubyte和train-labels-idx1-ubyte
    auto images = read_mnist_images(dir + "/data/FashionMNIST/raw/t10k-images-idx3-ubyte");
    // 读取测试集标签
    auto labels = read_mnist_labels(dir + "/data/FashionMNIST/raw/t10k-labels-idx1-ubyte");
    // 读取模型参数
    // std::cout << dir << std::endl;
    auto conv1_weight = read_param(dir + "/conv1.weight.txt");
    auto conv1_bias = read_param(dir + "/conv1.bias.txt");
    auto conv2_weight = read_param(dir + "/conv2.weight.txt");
    auto conv2_bias = read_param(dir + "/conv2.bias.txt");
    auto fc1_weight = read_param(dir + "/fc1.weight.txt");
    auto fc1_bias = read_param(dir + "/fc1.bias.txt");
    auto fc2_weight = read_param(dir + "/fc2.weight.txt");
    auto fc2_bias = read_param(dir + "/fc2.bias.txt");
    auto fc3_weight = read_param(dir + "/fc3.weight.txt");
    auto fc3_bias = read_param(dir + "/fc3.bias.txt");

    // 打印每一个标签，仅用于调试！
    
    // for (const auto& label : labels) {
    //     std::cout << label << " ";
    // }
    // std::cout<<std::endl;
    

    // 开始计时，使用chrono计时，不支持其它计时方式
    auto start = std::chrono::high_resolution_clock::now();

    // 进行推理
    std::cout << images.size() << std::endl; // 1w张图
    std::cout << images[0].size() << std::endl; // 28 * 28 = 784

    // 参数加载
    // std::cout << fc3_bias.size() << std::endl;

    for (int t = 0; t < images.size(); t++) {
        // TODO ...在这里实现利用CUDA对图片进行深度学习的推理过程，当然，你也可以改进for循环以使用batch推理提速...

        // 打印每一张图片，仅用于调试！
        
        // for(int i = 0;i <28; i++)
        // {
        //     for(int j = 0; j<28; j++)
        //     {
        //         std::cout << (float(images[t][i*28 + j])>0.5?1:0);
        //     }
        //     std::cout << std::endl;
        // }
        // std::cout << std::endl;

        //conv2d1 input_channal = 1, output_channal = 6, kernel_size = 5 
        for (int )

    }


    // 向主机端同步以等待所有异步调用的GPU kernel执行完毕，这句必须要有
    hipDeviceSynchronize();

    // 结束计时
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;

    // 输出结果，请严格保持此输出格式，并把0.0001替换成实际的准确率，请不要输出除了此结果之外的任何内容！！！
    std::cout << std::fixed << std::setprecision(2) << diff.count() << ":0.0001";

    return 0;
}
